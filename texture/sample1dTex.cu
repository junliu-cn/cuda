#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.h"

// kernel
__global__ void texture1d_kernel(float* output, 
                                 hipTextureObject_t texObj, 
                                 float shift, 
                                 int n)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n) {
    // fetch original value
    output[i] = tex1D<float>(texObj, i + shift);
  }
}

// Host code
int main()
{
  // input
  const int n = 8;
  const int nsize = sizeof(float) * n;
  float *h_input = new float[n];
  for (int i=0; i<n; ++i) {
    h_input[i] = i+1;
  }

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray *cuArray;
  CUDA_CHECK(hipMallocArray(&cuArray, &channelDesc, n, 1));
  //CUDA_CHECK(hipMemcpyToArray(cuArray, 0, 0, h_input, nsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy2DToArray(cuArray, 0, 0, h_input, nsize, nsize, 1, hipMemcpyHostToDevice));

  // output
  const int m = n;
  const int msize = sizeof(float) * m;
  float *h_output = new float[m];

  float *d_output = nullptr;
  CUDA_CHECK(hipMalloc(&d_output, msize));


  // texture resource
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // texture description
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;

  hipTextureObject_t texObj = 0;
  CUDA_CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

  int blockSize = 16;
  int gridSize = (n + blockSize - 1) / blockSize;

  float shift = 0.5f; // fetch original value

  texture1d_kernel<<<gridSize, blockSize>>>(d_output, texObj, shift, n);

  CUDA_CHECK(hipMemcpy(h_output, d_output, msize, hipMemcpyDeviceToHost));

  for (int i=0; i<m; ++i) {
    printf("%.1f ", h_output[i]);
  }
  printf("\n");

  // free resource
  hipDestroyTextureObject(texObj);
  hipFreeArray(cuArray);
  hipFree(d_output);

  delete[] h_input;
  delete[] h_output;

  return 0;
}
