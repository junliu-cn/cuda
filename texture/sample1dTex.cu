#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <string>
#include <list>

#include "common.h"

// kernel
__global__ void texture1d_kernel(float* output, 
                                 hipTextureObject_t texObj, 
                                 float shift, 
                                 int n,
                                 int padding)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n + 2*padding) 
  {
    // fetch original value
    output[i] = tex1D<float>(texObj, (float)(i + shift - padding)/n);
  }
}


const std::list<std::pair<hipTextureAddressMode, std::string>> kAddressMode = 
   {{hipAddressModeWrap, "wrap"},
    {hipAddressModeClamp, "clamp"}, 
    {hipAddressModeMirror, "mirror"},  // mirror mode only supported for normalized coordinates
    {hipAddressModeBorder, "border"}}; // border(period) mode only supported for normalized coordinates

void printResult(const std::string &str, float *out, int n, int padding);

// Host code
int main()
{
  // input
  const int n = 4;
  const int nsize = sizeof(float) * n;
  float *h_input = new float[n];
  for (int i=0; i<n; ++i) 
  {
    h_input[i] = i+1;
  }

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  hipArray *cuArray;
  CUDA_CHECK(hipMallocArray(&cuArray, &channelDesc, n, 1));
  //CUDA_CHECK(hipMemcpyToArray(cuArray, 0, 0, h_input, nsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy2DToArray(cuArray, 0, 0, h_input, nsize, nsize, 1, hipMemcpyHostToDevice));

  // output
  int padding = 4;
  const int m = n + 2*padding;
  const int msize = sizeof(float) * m;
  float *h_output = new float[m];

  float *d_output = nullptr;
  CUDA_CHECK(hipMalloc(&d_output, msize));


  // texture resource
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // texture description
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  //texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t texObj = 0;

  int blockSize = 16;
  int gridSize = (n + blockSize - 1) / blockSize;

  float shift = 0.5f; // +0.5f original value

  for (const auto &mode: kAddressMode) {
    texDesc.addressMode[0] = mode.first;
    CUDA_CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
    texture1d_kernel<<<gridSize, blockSize>>>(d_output, texObj, shift, n, padding);

    CUDA_CHECK(hipMemcpy(h_output, d_output, msize, hipMemcpyDeviceToHost));

    printResult(mode.second, h_output, m, padding);
  }

  // free resource
  hipDestroyTextureObject(texObj);
  hipFreeArray(cuArray);
  hipFree(d_output);

  delete[] h_input;
  delete[] h_output;

  return 0;
}


void printResult(const std::string &str, float *h_output, int m, int padding) {

  printf("%6s mode: ", str.c_str());

  for (int i=0; i<m; ++i) 
  {
    printf("%.1f", h_output[i]);

    if (i == padding-1) {
      printf(" | ");
    }
    else if (i == m - padding -1) {
      printf(" | ");
    } else {
      printf(" ");
    }
  }
  printf("\n");
}

